#include "hip/hip_runtime.h"
// outdependent and outinvariant are just generic computations

__global__ void gatherKernel(int *in, int* out, int num_in, int num_out) {

    int outIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if( outIdx < num_out) {
        for(unsigned int inIdx = 0; inIdx < num_in; ++inIdx) {
            atomicAdd(
                    &(out[outIdx]),
                    outDependent(outInvariant(in[inIdx]), inIdx, outIdx)
                    );
        }
    }

}
