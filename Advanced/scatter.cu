#include "hip/hip_runtime.h"
// intermediate is just a generic computation, can be anything

__global void scatterKernel(int *in, int *out, int num_in, int num_out) {

    unsigned int inIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if( inIdx < num_in) {
        unsigned int intermediate = outInvariant(in[inIdx]);
        for( unsigned int outIdx = 0; outIdx < num_out; ++outIdx) {
            atomicAdd(
                    &(out[outIdx]),
                    outDependent(intermediate, inIdx, outIdx)
                    );
        }
    }

}
