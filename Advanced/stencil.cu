#include "hip/hip_runtime.h"
#define TILE_SIZE 32

__device__ unsigned int Idx3D(int nx, int ny, int i, int j, int k) {
    return k*nx*ny + j*nx + i;
}

__global__ void stencilKernel(int *A0, int *Anext, int nx, int ny, int nz) {

    __shared__ float ds_A[TILE_SIZE][TILE_SIZE];
    
    unsigned int tx = threadIdx.x;          unsigned int ty = threadIdx.y;
    unsigned int dx = blockDim.x;           unsigned int dy = blockDim.y;
    unsigned int  i = blockIdx.x * dx + tx; unsigned int  j = blcokIdx.y * dy + ty;

    if((i < nx) && (j < ny)) {

        float bottom = A0[Idx3D(nx, ny, i, j, 0)];
        float center = A0[Idx3D(nx, ny, i, j, 1)];
        float top    = A0[Idx3D(nx, ny, i, j, 2)];

        for(int k = 1; k < nz-1; k++) {
            ds_A[ty][tx] = cewnter;
            __syncthreads();
            Anext[Idx3D(nx, ny, i, j, k)] = bottom + top - 6 * center + 
                ((tx>0)    ? ds_A[ty][tx-1] : (i==0)    ? 0 : A0[Idx3d(nx,ny,i-1,j,k)]) +
                ((tx<dx-1) ? ds_A[ty][tx+1] : (i==nx-1) ? 0 : A0[Idx3d(nx,ny,i+1,j,k)]) +
                ((ty>0)    ? ds_A[ty-1][tx] : (i==0)    ? 0 : A0[Idx3d(nx,ny,i,j-1,k)]) +
                ((ty<dy-1) ? ds_A[ty+1][tx] : (i==ny-1) ? 0 : A0[Idx3d(nx,ny,i,j+1,k)]);
            bottom = center;
            center = top;
            __syncthreads();
            if(k + 2 < nz) top = A0[Idx3D(nx, ny, i, j, k+2)];
        }
    }

}
