#include "hip/hip_runtime.h"
__global__ void convolutionKernel(Matrix M, Matrix N, Matrix P) {

    __shared__ float N_s[BLOCK_SIZE][BLOCK_SIZE];

    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int n = KERNEL_SIZE / 2;

    int row_o = by * TILE_SIZE + ty; int row_i = row_o - n;
    int col_o = bx * TILE_SIZE + tx; int col_i = col_o - n;

    float output = 0.0;

    if ((row_i >= 0) && (row_i < N.height) &&
        (col_i >= 0) && (col_i < N.width) ) {
        N_s[ty][tx] = N.elements[row_i * N.width + col_i];
    } else {
        N_s[ty][tx] = 0.0;
    }

    __syncthreads();

    if ((ty < TILE_SIZE) && (tx < TILE_SIZE)) {
        for (int i = 0; i < KERNEL_SIZE; i++) {
            for (int j = 0; j < KERNEL_SIZE; j++) {
                output += M_c[i][j] * N_s[i+ty][j+tx];
            }
            __syncthreads();
        }
        if ((row_o < P.height) && (col_o < P.width) ) {
            P.elements[row_o * P.width + col_o] = output;
        }
    }

}
