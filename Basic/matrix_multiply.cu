#include "hip/hip_runtime.h"
__global__ void matrixMultiplyKernel(Matrix M, Matrix N, Matrix P) {

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    float pval = 0;
    for (int k = 0; k < MATRIX_SIZE; k++) {
        pval += 
            M.elements[row * MATRIX_SIZE + k] * 
            N.elements[k * MATRIX_SIZE + col];
    }
    P.elements[row * MATRIX_SIZE + col] = pval;
}
