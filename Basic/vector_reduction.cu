#include "hip/hip_runtime.h"
__global__ void reductionKernel(float *g_data, int n) {

    __shared__ float partialSum[2*BLOCK_SIZE];

    int tx = threadIdx.x;
    int bx = blockDim.x
    int start = 2 * blockIdx.x * bx;

    // Setting the identity for addition as well as corner turning
    partialSum[tx] = ((start + tx) < n) ? g_data[start + tx] : 0;
    partialSum[bx + tx] = ((start + bx + t) < n) ? g_data[start + bx + tx] : 0;

    for (int stride = bx; stride >= 1; stride >>= 1) {
        __syncthreads();
        if (tx < stride) partialSum[tx] += partialSum[tx + stride];
    }
    if (tx == 0) g_data[blockIdx.x] = partialSum[0];
}
