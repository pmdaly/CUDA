#include "hip/hip_runtime.h"
__global__ void matrixMultiplyTiledKernel(Matrix M, Matrix N, Matrix P) {

    __shared__ float M_s[TILE_WIDTH][TILE_WIDTH];
    __shared__ float N_s[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    float pval = 0;
    for (int tile_idx = 0; tile_idx < ceil((float)M.width / TILE_WIDTH);
            ++tile_idx) {

        // Validate indices are w/in M
        if (row < M.height && tile_idx * TILE_WIDTH + tx < M.width) {
            M_s[ty][tx] = 
                M.elements[row * M.width + tile_idx * TILE_WIDTH + tx];
        } else {
            M_s[ty][tx] = 0.0;
        }

        // Validate indices are w/in N
        if (col < N.width && tile_idx * TILE_WIDTH + ty < N.height) {
            N_s[ty][tx] = 
                N.elements[(tile_idx * TILE_WIDTH + ty) * N.width + col];
        } else {
            N_s[ty][tx] = 0.0;
        }

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k) pval += M_s[ty][k] * N_s[k][tx];

        __syncthreads();

    }

    if (row < M.height && col < N.width) {
        P.elements[row * N.width + col] = pval;
    }

}
