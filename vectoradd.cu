
#include <hip/hip_runtime.h>
__global__ void vectorAddKernel(float *A, float *B, float *C, int n) {

    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < n) C[i] = A[i] + B[i];

}
